#include "hip/hip_runtime.h"
#include "fix_cpu.cuh"
#include "image.hh"

#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include <thrust/transform.h>
#include <rmm/device_uvector.hpp>
#include <hipcub/hipcub.hpp>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    }

__global__ void apply_pixel_transformation(int* buffer, int image_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < image_size) {
        if (idx % 4 == 0)
            buffer[idx] += 1;
        else if (idx % 4 == 1)
            buffer[idx] -= 5;
        else if (idx % 4 == 2)
            buffer[idx] += 3;
        else if (idx % 4 == 3)
            buffer[idx] -= 8;
    }
}

__global__ void histogram_kernel(int* buffer, int image_size, int* histogram) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < image_size) {
        atomicAdd(&histogram[buffer[idx]], 1);
    }
}

__global__ void equalize_histogram(int* buffer, int image_size, int* histogram, int cdf_min) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < image_size) {
        float normalized = ((histogram[buffer[idx]] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f;
        buffer[idx] = roundf(normalized);
    }
}

void fix_image_gpu(Image& to_fix) {
    const int image_size = to_fix.width * to_fix.height;
    constexpr int garbage_val = -27;

    // Allocate device memory using RMM for better memory management
    rmm::device_uvector<int> d_buffer(to_fix.size(), rmm::cuda_stream_default);
    rmm::device_uvector<int> d_predicate(to_fix.size(), rmm::cuda_stream_default);
    rmm::device_uvector<int> d_histogram(256, rmm::cuda_stream_default);

    // Copy buffer from host to device
    //hipMemcpy(d_buffer.data(), to_fix.buffer, sizeof(int) * to_fix.size(), hipMemcpyHostToDevice);
    std::cout << "Checkpoint 1" << std::endl;

    thrust::positive<int> op;
    // #1 Compact - Build predicate vector
    thrust::transform(d_buffer.begin(), d_buffer.end(), d_predicate.begin(), op);
    std::cout << "Checkpoint 2" << std::endl;

    // Compute the exclusive sum of the predicate (compact step)
    thrust::exclusive_scan(d_predicate.begin(), d_predicate.end(), d_predicate.begin());
    std::cout << "Checkpoint 3" << std::endl;

    // Scatter to the corresponding addresses
    thrust::for_each(thrust::make_counting_iterator(0), thrust::make_counting_iterator(to_fix.size()), [d_buffer = d_buffer.data(), d_predicate = d_predicate.data(), garbage_val] __device__(int idx) {
        if (d_buffer[idx] != garbage_val) {
            d_buffer[d_predicate[idx]] = d_buffer[idx];
        }
    });
    std::cout << "Checkpoint 4" << std::endl;

    // #2 Apply map to fix pixels
    const int block_size = 256;
    int grid_size = (image_size + block_size - 1) / block_size;
    apply_pixel_transformation<<<grid_size, block_size>>>(d_buffer.data(), image_size);
    std::cout << "Checkpoint 5" << std::endl;

    // #3 Histogram equalization
    // Histogram initialization (use thrust to zero the histogram)
    thrust::fill(d_histogram.begin(), d_histogram.end(), 0);
    std::cout << "Checkpoint 6" << std::endl;

    // Calculate histogram
    histogram_kernel<<<grid_size, block_size>>>(d_buffer.data(), image_size, d_histogram.data());
    std::cout << "Checkpoint 7" << std::endl;

    // Compute the inclusive sum scan of the histogram
    thrust::inclusive_scan(d_histogram.begin(), d_histogram.end(), d_histogram.begin());
    std::cout << "Checkpoint 8" << std::endl;

    // Find the first non-zero value in the cumulative histogram (on device)
    int cdf_min;
    thrust::device_ptr<int> dev_histogram_ptr = thrust::device_pointer_cast(d_histogram.data());
    auto first_non_zero = thrust::find_if(dev_histogram_ptr, dev_histogram_ptr + 256, [] __device__(int v) {
        return v != 0;
    });
    hipMemcpy(&cdf_min, thrust::raw_pointer_cast(first_non_zero), sizeof(int), hipMemcpyDeviceToHost);

    // Apply histogram equalization transformation
    equalize_histogram<<<grid_size, block_size>>>(d_buffer.data(), image_size, d_histogram.data(), cdf_min);
    std::cout << "Checkpoint 9" << std::endl;

    // Copy the buffer back to host
    hipMemcpy(to_fix.buffer, d_buffer.data(), sizeof(int) * to_fix.size(), hipMemcpyDeviceToHost);
}