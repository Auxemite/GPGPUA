#include "hip/hip_runtime.h"
#include "fix_cpu.cuh"
#include "image.hh"

#include <string>
#include <iostream>
#include <thrust/device_vector.h>
#include <thrust/async/scan.h>
#include <thrust/async/transform.h>
#include <thrust/remove.h>
// #include <raft/stats/histogram.cuh>
// #include <raft/matrix/matrix_view.hpp>
#include <rmm/device_uvector.hpp>
#include <hipcub/hipcub.hpp>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    }

void print_log(const std::string& message) {
    bool debug = false;
    if (debug)
        std::cout << message << std::endl;
}

__global__ void apply_pixel_transformation(int* buffer, int image_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < image_size) {
        if (idx % 4 == 0)
            buffer[idx] += 1;
        else if (idx % 4 == 1)
            buffer[idx] -= 5;
        else if (idx % 4 == 2)
            buffer[idx] += 3;
        else if (idx % 4 == 3)
            buffer[idx] -= 8;
    }
}

__global__ void histogram_kernel(int* buffer, int image_size, int* histogram) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < image_size) {
        atomicAdd(&histogram[buffer[idx]], 1);
    }
}

__global__ void equalize_histogram(int* buffer, int image_size, int* histogram, int cdf_min) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < image_size) {
        float normalized = ((histogram[buffer[idx]] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f;
        buffer[idx] = roundf(normalized);
    }
}

struct is_negate_27
{
  __host__ __device__
  bool operator()(const int x)
  {
    return x == -27;
  }
};

void fix_image_gpu(rmm::device_uvector<int>& d_buffer, const int image_size) {
    // raft::resources handle;
    // Allocate device memory using thurst
    rmm::device_uvector<int> d_histogram(256, d_buffer.stream());
    hipMemsetAsync(d_histogram.data(), 0, sizeof(int) * 256, d_buffer.stream());
    hipStreamSynchronize(d_buffer.stream());
    print_log("Checkpoint 1");

    // #1 Compact - Build predicate vector
    thrust::remove_if(thrust::cuda::par.on(d_buffer.stream()), d_buffer.begin(), d_buffer.end(), is_negate_27());
    hipStreamSynchronize(d_buffer.stream());
    print_log("Checkpoint 2");
    
    // #2 Apply map to fix pixels
    const int block_size = 256;
    int grid_size = (image_size + block_size - 1) / block_size;
    apply_pixel_transformation<<<grid_size, block_size, 0, d_buffer.stream()>>>(d_buffer.data(), image_size);
    hipStreamSynchronize(d_buffer.stream());
    print_log("Checkpoint 3");

    // #3 Histogram equalization
    // Calculate histogram
    // raft::device_matrix_view<const int, int, raft::col_major> data_view(d_buffer.data().get(), image_size, 1);
    // raft::device_matrix_view<int, int, raft::col_major> bins_view(d_histogram.data().get(), 256, 1);
    // raft::stats::histogram<int, int>(handle, raft::stats::HistType::BASIC, data_view, bins_view);
    histogram_kernel<<<grid_size, block_size, 0, d_buffer.stream()>>>(d_buffer.data(), image_size, d_histogram.data());
    hipStreamSynchronize(d_buffer.stream());
    print_log("Checkpoint 4");

    // Compute the inclusive sum scan of the histogram
    thrust::async::inclusive_scan(thrust::cuda::par.on(d_buffer.stream()), d_histogram.begin(), d_histogram.end(), d_histogram.begin());
    hipStreamSynchronize(d_buffer.stream());
    print_log("Checkpoint 5");

    // Find the first non-zero value in the cumulative histogram (on device)
    int cdf_min;
    auto first_non_zero = thrust::find_if(thrust::cuda::par.on(d_buffer.stream()), d_histogram.begin(), d_histogram.end(), [] __device__(int v) {
        return v != 0;
    });
    hipMemcpyAsync(&cdf_min, thrust::raw_pointer_cast(&(*first_non_zero)), sizeof(int), hipMemcpyDeviceToHost, d_buffer.stream());
    hipStreamSynchronize(d_buffer.stream());
    print_log("Checkpoint 6");

    // Apply histogram equalization transformation
    equalize_histogram<<<grid_size, block_size, 0, d_buffer.stream()>>>(d_buffer.data(), image_size, d_histogram.data(), cdf_min);
    hipStreamSynchronize(d_buffer.stream());
    print_log("Checkpoint 7");
}