#include "hip/hip_runtime.h"
#include "kernel.cuh"

#include <raft/core/device_span.hpp>
#include <rmm/device_uvector.hpp>
#include <algorithm>
template <typename T>
__global__
void kernel_map(raft::device_span<T> buffer)
{
    unsigned int tid = threadIdx.x;
    unsigned int mod = blockIdx.x%4;
    unsigned int i = int(blockIdx.x/4)*blockDim.x*4+mod+threadIdx.x*4;

    if(i>=buffer.size())
        return ;
    if(mod==0)
    {
        buffer[i]+=1;
    }
    else if(mod==1)
    {
        buffer[i]-=5;
    }
    else if(mod==2)
    {
        buffer[i]+=3;
    }
    else
    {
        buffer[i]-=8;
    }

}

template <typename T>
__global__
void kernel_map_2(raft::device_span<T> buffer)
{
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x+tid;
    unsigned int mod = i%4;

    if(i>=buffer.size())
        return ;
    if(mod==0)
    {
        buffer[i]+=1;
    }
    else if(mod==1)
    {
        buffer[i]-=5;
    }
    else if(mod==2)
    {
        buffer[i]+=3;
    }
    else
    {
        buffer[i]-=8;
    }

}


void map_modulo(rmm::device_uvector<int>& buffer)
{
    unsigned int min_block =(buffer.size()+4-1)/4;
    int taille_block =(min_block+32-1)/32;
    int t = std::min(taille_block*32,1024);
    int nb_block = (buffer.size()+t-1)/t;
    if(nb_block%4!=0)
    {
        nb_block+=4-(nb_block%4);
    }
    kernel_map<int><<<nb_block,t,0,buffer.stream()>>>(raft::device_span<int>(buffer.data(),buffer.size()));
    //kernel_map_2<int><<<nb_block,t,0,buffer.stream()>>>(raft::device_span<int>(buffer.data(),buffer.size()));
}
