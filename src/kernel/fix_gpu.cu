#include "hip/hip_runtime.h"
#include "fix_gpu.cuh"
#include "kernel.cuh"

#include <array>
#include <numeric>
#include <algorithm>
#include <cmath>
#include <thrust/async/scan.h>

#define CUDA_CHECK(call) \
    { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    }

void fix_image_gpu(Image& to_fix,hipStream_t& stream)
{

    const int image_size = to_fix.width * to_fix.height;

    rmm::device_uvector<int> d_buffer(to_fix.size(),stream);
    
    hipMemcpyAsync(d_buffer.data(),to_fix.buffer,to_fix.size()*sizeof(int),hipMemcpyHostToDevice,stream); 
    
    hipStreamSynchronize(stream);
    // #1 Compact

    // Build predicate vector
    rmm::device_uvector<int> predicate(d_buffer.size()+1,stream); 
    
    creation_mask(d_buffer,predicate);
    
    hipStreamSynchronize(stream);
    // Compute the exclusive sum of the predicate

    //thrust::async::inclusive_scan(thrust::cuda::par.on(stream),predicate.begin(), predicate.end(), predicate.begin());
    DecoupledLookBack_Scan(predicate);
    
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Scatter to the corresponding addresses
    rmm::device_uvector<int> res(image_size,stream);
   

    scatter(d_buffer,predicate,res); 
    
    CUDA_CHECK(hipStreamSynchronize(stream));

    // #2 Apply map to fix pixels

    map_classique(res,image_size);

    hipStreamSynchronize(stream);
    

    // #3 Histogram equalization

    // Histogram

    rmm::device_uvector<int> histo(256,stream);

    hipMemsetAsync(histo.data(),0,histo.size()*sizeof(int),stream);

    hipStreamSynchronize(stream);

    fill_histo(res,histo);
    
    hipStreamSynchronize(stream);
    

    // Compute the inclusive sum scan of the histogram

    Scan_histo(histo);

    hipStreamSynchronize(stream);
    // Find the first non-zero value in the cumulative histogram
    const int found = find_first_value(histo);

    // Apply the map transformation of the histogram equalization
    hipStreamSynchronize(stream);

    last_mapping(res,histo,found);
    /*std::transform(to_fix.buffer, to_fix.buffer + image_size, to_fix.buffer,
        [image_size, cdf_min, &histo](int pixel)
            {
                return std::roundf(((histo[pixel] - cdf_min) / static_cast<float>(image_size - cdf_min)) * 255.0f);
            }
    );*/

    hipStreamSynchronize(stream);

    hipMemcpyAsync(to_fix.buffer,res.data(),image_size*sizeof(int),hipMemcpyDeviceToHost,stream);
    
    hipStreamSynchronize(stream);
}
